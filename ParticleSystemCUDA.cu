#include "hip/hip_runtime.h"
//
// Created by Eric Fang on 5/10/17.
//

#include "ParticleSystemCUDA.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <random>

void update(int gridSize, int particleCount, int iterations, float3 *velocity,
            float3 *position_next, float3 *position, int *neighbor_counts,
            int *neighbors, int *grid_counts, int *grid, float *lambda);
void initialize(struct systemParams *p);

#define cudaCheck(x)                                                          \
  {                                                                           \
    hipError_t err = x;                                                      \
    if (err != hipSuccess) {                                                 \
      printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); \
      assert(0);                                                              \
    }                                                                         \
  }

inline float3 operator+(float3 a, float3 b) {
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

ParticleSystemCUDA::ParticleSystemCUDA(unsigned numParticles,
                                       glm::vec3 bounds_max, std::string config)
    : ParticleSystem(numParticles, bounds_max) {
  systemParams params;
  params.poly6_const = poly6_const;
  params.spiky_const = spiky_const;
  params.maxGridCount = maxNeighbors;
  params.maxNeighbors = maxNeighbors;
  params.particleCount = numParticles;
  params.bounds_min = make_float3(bounds_min.x, bounds_min.y, bounds_min.z);
  params.bounds_max = make_float3(bounds_max.x, bounds_max.y, bounds_max.z);
  params.gravity = make_float3(gravity.x, gravity.y, gravity.z);
  params.dist_from_bound = dist_from_bound;
  params.delta_q = delta_q;
  params.iterations = iterations;
  params.c = c;
  params.k = k;
  params.epsilon = epsilon;
  params.rest_density = rest_density;
  params.dt = dt;
  params.gridX = int(ceil((bounds_max.x - bounds_min.x) / h));
  params.gridY = int(ceil((bounds_max.y - bounds_min.y) / h));
  params.gridZ = int(ceil((bounds_max.z - bounds_min.z) / h));
  params.h = h;
  float thickness = 0.1;
  hostParticlePos = (float3 *)malloc(numParticles * sizeof(float3));
  std::default_random_engine generator;

  if (config == "dam") {
    std::uniform_real_distribution<float> distributionX(bounds_min.x + 0.1,
                                                        bounds_min.x + 40);
    std::uniform_real_distribution<float> distributionY(bounds_min.y + 0.1,
                                                        bounds_max.y - 0.1);
    std::uniform_real_distribution<float> distributionZ(bounds_min.z + 0.1,
                                                        bounds_max.z - 0.1);
    for (int i = 0; i < numParticles; i++) {
      hostParticlePos[i] =
          make_float3(distributionX(generator), distributionY(generator),
                      distributionZ(generator));
    }
  } else if (config == "sphere") {
    float r = std::min(std::min(bounds_max.x - bounds_min.x,
                                bounds_max.y - bounds_min.y),
                       bounds_max.z - bounds_min.z) /
              2.0;
    float3 offset = make_float3((bounds_max.x - bounds_min.x) / 2.0,
                                (bounds_max.y - bounds_min.y) / 2.0,
                                (bounds_max.z - bounds_min.z) / 2.0);
    std::uniform_real_distribution<float> distributionR(-r, r);
    float x, y, z;
    for (int i = 0; i < numParticles; i++) {
      do {
        x = distributionR(generator);
        y = distributionR(generator);
        z = distributionR(generator);
      } while (x * x + y * y + z * z >= r * r);
      hostParticlePos[i] = make_float3(x, y, z) + offset;
    }
  } else {
    std::uniform_real_distribution<float> distribution(bounds_min.x + 5,
                                                       bounds_max.x - 5);

    for (int i = 0; i < numParticles; i++) {
      hostParticlePos[i] =
          make_float3(distribution(generator), distribution(generator),
                      distribution(generator));
    }
  }

  gridSize = params.gridX * params.gridY * params.gridZ;

  cudaCheck(hipMalloc((void **)&particlePos, numParticles * sizeof(float3)));
  cudaCheck(hipMalloc((void **)&particleVel, numParticles * sizeof(float3)));
  cudaCheck(
      hipMalloc((void **)&particlePosNext, numParticles * sizeof(float3)));
  cudaCheck(hipMalloc((void **)&particleLambda, numParticles * sizeof(float)));
  cudaCheck(hipMalloc((void **)&neighborCounts, numParticles * sizeof(int)));
  cudaCheck(hipMalloc((void **)&neighbors,
                       numParticles * maxNeighbors * sizeof(int)));
  cudaCheck(hipMalloc((void **)&gridCount, gridSize * sizeof(int)));
  cudaCheck(
      hipMalloc((void **)&grid, gridSize * params.maxGridCount * sizeof(int)));

  cudaCheck(hipMemset(particlePos, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(particleVel, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(particlePosNext, 0, numParticles * sizeof(float3)));
  cudaCheck(hipMemset(particleLambda, 0, numParticles * sizeof(float)));
  cudaCheck(hipMemset(neighborCounts, 0, numParticles * sizeof(int)));
  cudaCheck(
      hipMemset(neighbors, 0, numParticles * maxNeighbors * sizeof(int)));
  cudaCheck(hipMemset(gridCount, 0, gridSize * sizeof(int)));
  cudaCheck(hipMemset(grid, 0, gridSize * params.maxGridCount * sizeof(int)));

  cudaCheck(hipMemcpy(particlePos, hostParticlePos,
                       numParticles * sizeof(float3), hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(particlePosNext, hostParticlePos,
                       numParticles * sizeof(float3), hipMemcpyHostToDevice));
  initialize(&params);
}

ParticleSystemCUDA::~ParticleSystemCUDA() {
  cudaCheck(hipFree(particlePos));
  cudaCheck(hipFree(particleVel));
  cudaCheck(hipFree(particlePosNext));
  cudaCheck(hipFree(particleLambda));
  cudaCheck(hipFree(neighborCounts));
  cudaCheck(hipFree(neighbors));
  cudaCheck(hipFree(gridCount));
  cudaCheck(hipFree(grid));
  free(hostParticlePos);
}

float *ParticleSystemCUDA::getParticlePos() {
#ifdef DEVICE_RENDER
  return (float *)particlePos;

#else
  cudaCheck(hipMemcpy(hostParticlePos, particlePos,
                       numParticles * sizeof(float3), hipMemcpyDeviceToHost));
  return &hostParticlePos[0].x;

#endif /* DEVICE_RENDER */
}

void ParticleSystemCUDA::step() {
  update(gridSize, numParticles, iterations, particleVel, particlePosNext,
         particlePos, neighborCounts, neighbors, gridCount, grid,
         particleLambda);
}
